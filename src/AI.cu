#include "hip/hip_runtime.h"
#include "AI.cuh"

std::random_device dev;
std::mt19937 rng(dev());
std::uniform_int_distribution<std::mt19937::result_type> dist1000(0, 1000);


class MyException : public std::exception { 
private: 
    std::string message; 
  
public: 
    // Constructor accepts a const char* that is used to set 
    // the exception message 
    MyException(const char* msg) 
        : message(msg) 
    { 
    } 
  
    // Override the what() method to return our message 
    const char* what() const throw() 
    { 
        return message.c_str(); 
    } 
};

// void add(double* a, double* b, double* c, int N)
// {    
//     int threads = 256;
//     int blocks = (N + threads - 1) / threads;
//     addKernel<<<blocks, threads>>>(a, b, c, N);
//     hipDeviceSynchronize();
// }

int Perceptron::InitCuda()
{
    hipMallocManaged(&neuronsConfig, layers*sizeof(int));
    // hipMallocManaged(&neurons, layers*neuronsConfig[0]*sizeof(double));
    // hipMallocManaged(&weights, layers*neuronsConfig[0]*(neuronsConfig[0]-1)*sizeof(double));
    std::cout << "Cuda was initialized" << std::endl; 
    return 0;
}

int Perceptron::Init()
{
    // hipMallocManaged(&neuronsConfig, layers*sizeof(int));

    hipMallocManaged(&neurons, layers*neuronsConfig[0]*sizeof(double));
    // std::cout << layers*neuronsConfig[0] << std::endl;
    for (int i = 0; i < layers; i++)
    {
        for (int j = 0; j < neuronsConfig[0]; j++)
        {
            // std::cout << i*neuronsConfig[0]+j << std::endl;
            // std::cout << dist1000(rng) * 1.0 / 1000 << std::endl;
            neurons[i*neuronsConfig[0]+j] = dist1000(rng) * 1.0 / 1000;
            // std::cout << neurons[i*neuronsConfig[0]+j] << std::endl;
        }
    }
    std::cout << "Neurons were initialized" << std::endl;

    hipMallocManaged(&weights, layers*neuronsConfig[0]*(neuronsConfig[0]-1)*sizeof(double));
    // int lastIndex = 0;
    for (int i = 0; i < layers; i++)
    {
        for (int j = 0; j < neuronsConfig[i]; j++)
        {
            for (int k = 0; k < neuronsConfig[i]-1; k++)
            {
                // if (i*neuronsConfig[i]*(neuronsConfig[i]-1)+j*(neuronsConfig[i]-1)+k - lastIndex != 1)
                // {
                //     std::cout << lastIndex << "->" << i*neuronsConfig[i]*(neuronsConfig[i]-1)+j*(neuronsConfig[i]-1)+k << std::endl;
                // }
                // lastIndex = i*neuronsConfig[i]*(neuronsConfig[i]-1)+j*(neuronsConfig[i]-1)+k;
                // std::cout << i*neuronsConfig[i]*(neuronsConfig[i]-1)+j*(neuronsConfig[i]-1)+k << std::endl;
                weights[i*neuronsConfig[i]*(neuronsConfig[i]-1)+j*(neuronsConfig[i]-1)+k] = dist1000(rng) * 1.0 / 1000;
            }
        }
    }
    std::cout << "Weights were initialized" << std::endl;
    return 0;
}

double Perceptron::Sigmoid(double input)
{
    return 1/(1+exp(-input));
}

double Perceptron::ReLU(double input)
{
    if (input > 0){return input;}
    else {return 0;}
}

double Perceptron::Tanh(double input)
{
    return tanh(input);
}

__global__
void CalculateNeuronsKernel(double* neurons, double* weights, int* neuronsConfig, int layers, int layer)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    // int index = threadIdx.x;
    // int stride = blockDim.x;

    for (int i = index; i < neuronsConfig[layer+1]; i+=stride)
    {
        neurons[neuronsConfig[0]*(layer+1)+i] = 0;
        for (int j = 0; j < neuronsConfig[layer]; j++)
        {
            neurons[neuronsConfig[0]*(layer+1)+i] += neurons[neuronsConfig[0]*layer+j] * weights[layer*neuronsConfig[0]*(neuronsConfig[0]-1)+j*(neuronsConfig[0]-1)+i];
        }
    }
}

int Perceptron::CalculateNeurons(ActivationFunction activationFunction)
{
    for (int i = 0; i < layers-1; i++)
    {
        // std::cout << i << std::endl;
        gpuThreads = 256;
        gpuBlocks = (neuronsConfig[i] + gpuThreads - 1) / gpuThreads;
        CalculateNeuronsKernel<<<gpuBlocks, gpuThreads>>>(neurons, weights, neuronsConfig, layers, i);
        hipDeviceSynchronize();
    }
    for (int i = neuronsConfig[0]*(layers-1); i < neuronsConfig[0]*layers; i++)
    {
        // std::cout << neurons[i] << std::endl;
        switch (activationFunction)
        {
            case ActivationFunction::Sigmoid:
                neurons[i] = Sigmoid(neurons[i]);
                break;
            case ActivationFunction::ReLU:
                neurons[i] = ReLU(neurons[i]);
                break;
            case ActivationFunction::Tanh:
                neurons[i] = Tanh(neurons[i]);
                break;
        
            default:
                break;
        }
        // std::cout << neurons[i] << std::endl;
    }
    
    std::cout << "Neurons were recalculated" << std::endl;
    return 0;
}

// __global__
// void MeanSquaredErrorKernel(double* neurons, int* neuronsConfig, double* rightAnswer, int layer, double output)
// {
//     int index = blockIdx.x * blockDim.x + threadIdx.x;
//     int stride = blockDim.x * gridDim.x;
//     for (int i = index; i < neuronsConfig[layer]; i+=stride)
//     {
//         output += pow(neurons[neuronsConfig[0]*layer+i] - rightAnswer[i], 2);
//     }
// }

double Perceptron::MeanSquaredError(int layer)
{
    double output = 0.0;
    // gpuThreads = 256;
    // gpuBlocks = (neuronsConfig[layer] + gpuThreads - 1) / gpuThreads;
    // MeanSquaredErrorKernel<<<gpuBlocks, gpuThreads>>>(neurons, neuronsConfig, rightAnswer, layer, output);
    // hipDeviceSynchronize();
    for (int i = 0; i < neuronsConfig[layer]; i++)
    {
        // std::cout << neurons[neuronsConfig[0]*layer+i] << " - " << rightAnswer[i] << " = " << neurons[neuronsConfig[0]*layer+i] - rightAnswer[i] << std::endl;
        // std::cout << output << " + " << pow(neurons[neuronsConfig[0]*layer+i] - rightAnswer[i], 2) << " = ";
        output += pow(neurons[neuronsConfig[0]*layer+i] - rightAnswer[i], 2);
        // std::cout << output << std::endl;
    }
    // std::cout << output << std::endl;
    // std::cout << neuronsConfig[layer] << std::endl;
    // std::cout << layer << std::endl;
    output /= neuronsConfig[layer];
    return output;
}

double Perceptron::MeanAbsoluteError(int layer)
{
    double output = 0.0;
    for (int i = 0; i < neuronsConfig[layers]; i++)
    {
        output += abs(neurons[neuronsConfig[0]*(layer)+i] - rightAnswer[i]);
    }
    output /= (1.0 * neuronsConfig[layer]);
    return output;
}

double Perceptron::HuberLoss(int layer/* double delta */)
{
    if (delta == -1)
    {
        throw MyException("You must set the delta variable to use HuberLoss!");
    }
    double output = 0.0;
    for (int i = 0; i < neuronsConfig[layer]; i++)
    {
        if (abs(neurons[neuronsConfig[0]*(layer)+i] - rightAnswer[i]) > delta)
        {
            output += delta * (abs(neurons[neuronsConfig[0]*(layer)+i] - rightAnswer[i]) - 0.5f * delta);
        }
        else
        {
            output += pow(neurons[neuronsConfig[0]*(layer)+i] - rightAnswer[i], 2);
        }
    }
    output /= neuronsConfig[layer];
    return output;
}

double Perceptron::BinaryCrossEntropyLoss(int layer/* double clip */)
{
    if (clip == -1)
    {
        throw MyException("You must set the clip variable to use BinaryCrossEntropyLoss!");
    }
    double output = 0.0;
    for (int i = 0; i < neuronsConfig[layer]; i++)
    {
        rightAnswer[i] = max(clip, min(rightAnswer[i], 1-clip));
        output += (neurons[neuronsConfig[0]*(layer)+i]*log10(rightAnswer[i]+clip)) + 
        (1-neurons[neuronsConfig[0]*(layer)+i]) + (1-rightAnswer[i]+clip);
    }
    output /= neuronsConfig[layer];
    return output;
}

double Perceptron::CategoricalCrossEntropyLoss(int layer)
{
    // Work in progress...
    double output = 0.0;
    for (int i = 0; i < neuronsConfig[layer]; i++)
    {
        // output += ;
    }
    output /= neuronsConfig[layer];
    return output;
}

int Perceptron::CalculateCost(CostFunction costFunction, int layer)
{
    switch (costFunction)
    {
        case CostFunction::MeanSquared:
            cost = MeanSquaredError(layer);
            break;
        case CostFunction::MeanAbsolute:
            cost = MeanAbsoluteError(layer);
            break;
        case CostFunction::Huber:
            cost = HuberLoss(layer);
            break;
        case CostFunction::BinaryCrossEntropy:
            cost = BinaryCrossEntropyLoss(layer);
            break;
        
        default:
            break;
    }
    return 0;
}

__global__
void CalculateErrorKernel(double* neurons, int* neuronsConfig, double* rightAnswer, int layers, int i, double* error)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    // int index = threadIdx.x;
    // int stride = blockDim.x;
    double tmp;

    for (int j = index; j < neuronsConfig[i]; j+=stride)
    {
        tmp = neurons[neuronsConfig[0]*(layers-1)+i];
        error[i*neuronsConfig[0]+j] = tmp*(1-tmp)*(rightAnswer[i]-tmp);
    }
}

__global__
void BackpropagationKernel(double* neurons, double* weights, int* neuronsConfig, double* error, double learningRate, int i, int j)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    // int index = threadIdx.x;
    // int stride = blockDim.x;

    for (int k = index; k < neuronsConfig[i+1]; k+=stride)
    {
        weights[i*neuronsConfig[i]*(neuronsConfig[i]-1)+j*(neuronsConfig[i]-1)+k] += 
        learningRate*neurons[neuronsConfig[0]*i+j]*error[k];
    }
}

int Perceptron::Backpropagation(CostFunction costFunction)
{
    if (learningRate == -1)
    {
        throw MyException("You must set learningRate to use Backpropagation!");
    }

    double* error = new double[neuronsConfig[0]*layers];
    // double tmp;
    for (int i = 0; i < layers; i++)
    {
        gpuThreads = 256;
        gpuBlocks = (neuronsConfig[i] + gpuThreads - 1) / gpuThreads;
        CalculateErrorKernel<<<gpuBlocks, gpuThreads>>>(neurons, neuronsConfig, rightAnswer, layers, i, error);
        // for (int j = 0; j < neuronsConfig[i]; j++)
        // {
        //     tmp = neurons[neuronsConfig[0]*(layers-1)+i];
        //     error[i*neuronsConfig[0]+j] = tmp*(1-tmp)*(rightAnswer[i]-tmp);
        // }
    }
    hipDeviceSynchronize();
    for (int i = layers-2; i > 0; i--)
    {
        for (int j = 0; j < neuronsConfig[i]; j++)
        {
            BackpropagationKernel<<<gpuBlocks, gpuThreads>>>(neurons, weights, neuronsConfig, error, learningRate, i, j);
            // for (int k = 0; k < neuronsConfig[i+1]; k++)
            // {
            //     weights[i*neuronsConfig[i]*(neuronsConfig[i]-1)+j*(neuronsConfig[i]-1)+k] += 
            //     learningRate*neurons[neuronsConfig[0]*i+j]*error[k];
            // }
        }
    }
    hipDeviceSynchronize();
    return 0;
}

int Perceptron::SimulatedAnnealing(ActivationFunction activationFunction, CostFunction costFunction)
{
    if (temperature == -1 || temperatureDecreaseRate == -1)
    {
        throw MyException("You must set temperature and temperatureDecreaseRate to use SimulatedAnnealing!");
    }
    
    Perceptron candidate = *this;
    // std::cout << candidate.temperature << std::endl;
    for (int i = 0; i < temperature; i++)
    {
        candidate.weights[(int)(dist1000(rng) * (layers*neuronsConfig[0]*(neuronsConfig[0]-1) / 1000))] = dist1000(rng) * 1.0 / 1000;
    }
    candidate.CalculateNeurons(activationFunction);
    
    CalculateCost(costFunction, layers-1);
    candidate.CalculateCost(costFunction, layers-1);

    if (cost < candidate.cost)
    {
        weights = candidate.weights;
    }
    else
    {
        double deltaCost = cost - candidate.cost;
        // std::cout << cost << "; " << candidate.cost << std::endl;
        // std::cout << "Delta cost: " << deltaCost << std::endl;
        if ((dist1000(rng) * 1.0 / 1000) > exp(deltaCost / temperature))
        {
            weights = candidate.weights;
        }
    }
    
    temperature *= temperatureDecreaseRate;
    std::cout << "Temperature: " << temperature << std::endl;
    // free(&candidate);
    // delete &candidate;
    return 0;
}

double Perceptron::Train(ActivationFunction activationFunction, CostFunction costFunction, LearningAlgorithm learningAlgorithm)
{
    // std::cout << "Hello from the training function" << std::endl;
    CalculateNeurons(activationFunction);
    switch (learningAlgorithm)
    {
        case LearningAlgorithm::Backpropagation:
            Backpropagation(costFunction);
            break;
        case LearningAlgorithm::SimulatedAnnealing:
            SimulatedAnnealing(activationFunction, costFunction);
            break;
        
        default:
            break;
    }
    // double cost = 0;
    CalculateCost(costFunction, layers-1);
    std::cout << "Error: " << cost << std::endl;
    return cost;
}

int Perceptron::SaveWeights(std::string fileName)
{
    std::fstream weightsFile;
    std::string path = "./weights/"+fileName+".csv";
    weightsFile.open(path, std::ios::out);
    weightsFile << layers << ",";
    for (int i = 0; i < layers; i++)
    {
        weightsFile << neuronsConfig[i] << ",";
    }
    for (int i = 0; i < layers; i++)
    {
        for (int j = 0; j < neuronsConfig[i]; j++)
        {
            for (int k = 0; k < neuronsConfig[i]-1; k++)
            {
                weightsFile << weights[i*neuronsConfig[i]*(neuronsConfig[i]-1)+j*(neuronsConfig[i]-1)+k] << ",";
            }
        }
    }
    weightsFile.close();
    weightsFile.open(path, std::ios::in);
    std::string weightsString;
    weightsFile >> weightsString;
    weightsString.pop_back();
    weightsFile.close();
    weightsFile.open(path, std::ios::out);
    weightsFile << weightsString;
    weightsFile.close();
    return 0;
}

int Perceptron::Free()
{
    hipFree(neurons);
    hipFree(weights);
    hipFree(neuronsConfig);
    return 0;
}
